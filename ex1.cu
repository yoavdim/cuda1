#include "hip/hip_runtime.h"
#include "ex1.h"

//#define MAX(X, Y) (((X) > (Y)) ? (X) : (Y))

__device__ void prefix_sum(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int increment;
    // trick: allow running multiple times (modulu) & skip threads (negative arr_size)
    // arr_size must be the same for all or __syncthreads will cause deadlock
    tid      = (arr_size > 0)? tid % arr_size : 0; 
    arr_size = (arr_size > 0)? arr_size       : -arr_size;

    for (int stride = 1; stride < arr_size; stride *= 2) {
        if (tid >= stride) {
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride) {
            arr[tid] += increment;
        }
        __syncthreads();
    }
    return; 
}

/**
 * Perform interpolation on a single image
 *
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @param in_img single input image, in global memory.
 * @param out_img single output buffer, in global memory.
 */
__device__ 
void interpolate_device(uchar* maps ,uchar *in_img, uchar* out_img);

/**
* map between a thread to its tile, total tile number is TILES_COUNT^2
*/
__global__ int get_tile_id(int index) {
    int line = index / IMG_WIDTH;
    int col  = index % IMG_WIDTH;
    line = line / TILE_HEIGHT; // round down
    col  = col / TILE_WIDTH; 
    return line * TILES_COUNT + col;
}


__global__ void process_image_kernel(uchar *all_in, uchar *all_out, uchar *maps) {
    __shared__ int histograms[IMG_TILES][256]; // shared memory, so each block allocating a different one for each image
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int tnum = blockDim.x;

    for (int i = tid; i < IMG_TILES*256; i += tnum) { // set zero
        ((int*) histograms)[i] = 0;
    }
    __syncthreads();

    for (int index = tid; index < IMG_SIZE; index += tnum) { // calc histograms
        atomicAdd(&(histograms[get_tile_id(index)][all_in[IMG_SIZE*bid + index]]), 1);
    }
    __syncthreads();

    // run prefix sum in each tile --- ASSUME: tnum  >= 256
    for (int run=0; run < (IMG_TILES/(tnum/256)+1); run++) { // enforce same amount of entries to prefix_sum
        int tile = (tid/256) + run*(tnum/256);
        if (tile >= IMG_TILES) 
            prefix_sum(NULL, -256);  // keep internal syncthread from blocking the rest
        else 
            prefix_sum(&(histograms[tile][0]), 256);
    }
    __syncthreads();

    // create map
    for (int i = tid; i < IMG_TILES*256; i += tnum) { 
        int cdf = ((int*) histograms)[i];
        map[MAP_SIZE*bid + i] = (uchar) ((((double)cdf)*255)/(TILE_WIDTH*TILE_HEIGHT)); // cast will round down
    }
    __syncthreads();
    
    interpolate_device(maps, all_in, all_out);
    return; 
}

/* Task serial context struct with necessary CPU / GPU pointers to process a single image */
struct task_serial_context {
    // TODO define task serial memory buffers
    uchar *d_image_in;
    uchar *d_image_out;
    uchar *d_maps; 
};

/* Allocate GPU memory for a single input image and a single output image.
 * 
 * Returns: allocated and initialized task_serial_context. */
struct task_serial_context *task_serial_init()
{
    auto context = new task_serial_context;

    hipMalloc((void**)&(context->d_image_in),  IMG_SIZE);
    hipMalloc((void**)&(context->d_image_out), IMG_SIZE);
    hipMalloc((void**)&(context->d_maps), MAP_SIZE);
    //TODO: allocate GPU memory for a single input image, a single output image, and maps

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void task_serial_process(struct task_serial_context *context, uchar *images_in, uchar *images_out)
{
    //TODO: in a for loop:
    //   1. copy the relevant image from images_in to the GPU memory you allocated
    //   2. invoke GPU kernel on this image
    //   3. copy output from GPU memory to relevant location in images_out_gpu_serial
    for (int i=0; i < N_IMAGES; i++) {
        hipMemcpy(context->d_image_in + i*IMG_SIZE, images_in + i*IMG_SIZE, IMG_SIZE, hipMemcpyHostToDevice);
        process_image_kernel<<<1,THREAD_NUM>>>(context->d_image_in, context->d_image_out, context->d_maps);  // 
        hipDeviceSynchronize();
        CUDA_CHECK(hipGetLastError());
        hipMemcpy(context->d_image_out + i*IMG_SIZE, images_out + i*IMG_SIZE, IMG_SIZE, hipMemcpyDeviceToHost);
    }
}

/* Release allocated resources for the task-serial implementation. */
void task_serial_free(struct task_serial_context *context)
{
    //TODO: free resources allocated in task_serial_init
    hipFree(context->d_image_in);
    hipFree(context->d_image_out);
    hipFree(context->d_maps);
    free(context);
}

/* Bulk GPU context struct with necessary CPU / GPU pointers to process all the images */
struct gpu_bulk_context {
    // TODO define bulk-GPU memory buffers
    uchar *d_image_in;
    uchar *d_image_out;
    uchar *d_maps;
};

/* Allocate GPU memory for all the input images, output images, and maps.
 * 
 * Returns: allocated and initialized gpu_bulk_context. */
struct gpu_bulk_context *gpu_bulk_init()
{
    auto context = new gpu_bulk_context;
    hipMalloc((void**)&(context->d_image_in),  IMG_SIZE*N_IMAGES);
    hipMalloc((void**)&(context->d_image_out), IMG_SIZE*N_IMAGES);
    hipMalloc((void**)&(context->d_maps), MAP_SIZE*N_IMAGES);
    //TODO: allocate GPU memory for all the input images, output images, and maps

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void gpu_bulk_process(struct gpu_bulk_context *context, uchar *images_in, uchar *images_out)
{
    //TODO: copy all input images from images_in to the GPU memory you allocated
    //TODO: invoke a kernel with N_IMAGES threadblocks, each working on a different image
    //TODO: copy output images from GPU memory to images_out
    hipMemcpy(context->d_image_in, images_in, IMG_SIZE*N_IMAGES, hipMemcpyHostToDevice);
    process_image_kernel<<<N_IMAGES,THREAD_NUM>>>(context->d_image_in, context->d_image_out, context->d_maps);  // 
    hipDeviceSynchronize();
    CUDA_CHECK(hipGetLastError());
    hipMemcpy(context->d_image_out, images_out, IMG_SIZE*N_IMAGES, hipMemcpyDeviceToHost);
}

/* Release allocated resources for the bulk GPU implementation. */
void gpu_bulk_free(struct gpu_bulk_context *context)
{
    //TODO: free resources allocated in gpu_bulk_init
    hipFree(context->d_image_in);
    hipFree(context->d_image_out);
    hipFree(context->d_maps);
    free(context);
}
